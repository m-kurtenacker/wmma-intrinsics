
#include <hip/hip_runtime.h>
#include <mma.h>
using namespace nvcuda;

typedef wmma::fragment<wmma::accumulator, 16, 16, 16, half> cuda_acc_datatype;
typedef wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> cuda_mat_a_datatype;
typedef wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::col_major> cuda_mat_b_datatype;

typedef         __half_raw f16;

extern "C" {

__device__ void cuda_load_matrix_sync_c(cuda_acc_datatype* fragment, f16* data, int stride) {
    wmma::load_matrix_sync(*fragment, (half*) data, stride, wmma::mem_row_major);
}

__device__ void cuda_store_matrix_sync(f16* data, cuda_acc_datatype fragment, int stride) {
    wmma::store_matrix_sync((half*) data, fragment, stride, wmma::mem_row_major);
}

__device__ void cuda_load_matrix_sync_a(cuda_mat_a_datatype* fragment, f16* data, int stride) {
    wmma::load_matrix_sync(*fragment, (half*) data, stride);
}

__device__ void cuda_load_matrix_sync_b(cuda_mat_b_datatype* fragment, f16* data, int stride) {
    wmma::load_matrix_sync(*fragment, (half*) data, stride);
}

__device__ void cuda_mma_sync(cuda_acc_datatype* acc, cuda_mat_a_datatype a, cuda_mat_b_datatype b, cuda_acc_datatype c) {
    wmma::mma_sync(*acc, a, b, c);
}

}
